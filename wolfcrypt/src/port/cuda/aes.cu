#include "hip/hip_runtime.h"
/* aes.c
 *
 * Copyright (C) 2006-2023 wolfSSL Inc.
 *
 * This file is part of wolfSSL.
 *
 * wolfSSL is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License, or
 * (at your option) any later version.
 *
 * wolfSSL is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA 02110-1335, USA
 */

/*

DESCRIPTION
This library provides the interfaces to the Advanced Encryption Standard (AES)
for encrypting and decrypting data. AES is the standard known for a symmetric
block cipher mechanism that uses n-bit binary string parameter key with 128-bits,
192-bits, and 256-bits of key sizes.

*/
#ifdef HAVE_CONFIG_H
    #include <config.h>
#endif

#include <wolfssl/wolfcrypt/settings.h>
#include <wolfssl/wolfcrypt/error-crypt.h>

#include <wolfssl/wolfcrypt/aes.h>

#ifdef WOLFSSL_AESNI
#include <wmmintrin.h>
#include <emmintrin.h>
#include <smmintrin.h>
#endif /* WOLFSSL_AESNI */

#include <wolfssl/wolfcrypt/cpuid.h>

#ifdef WOLF_CRYPTO_CB
    #include <wolfssl/wolfcrypt/cryptocb.h>
#endif

#ifdef WOLFSSL_SECO_CAAM
#include <wolfssl/wolfcrypt/port/caam/wolfcaam.h>
#endif

#ifdef WOLFSSL_IMXRT_DCP
    #include <wolfssl/wolfcrypt/port/nxp/dcp_port.h>
#endif
#if defined(WOLFSSL_SE050) && defined(WOLFSSL_SE050_CRYPT)
    #include <wolfssl/wolfcrypt/port/nxp/se050_port.h>
#endif

#if defined(WOLFSSL_AES_SIV)
    #include <wolfssl/wolfcrypt/cmac.h>
#endif /* WOLFSSL_AES_SIV */

#if defined(WOLFSSL_HAVE_PSA) && !defined(WOLFSSL_PSA_NO_AES)
    #include <wolfssl/wolfcrypt/port/psa/psa.h>
#endif

#if defined(WOLFSSL_TI_CRYPT)
    #include <wolfcrypt/src/port/ti/ti-aes.c>
#else

#include <wolfssl/wolfcrypt/logging.h>

#ifdef NO_INLINE
    #include <wolfssl/wolfcrypt/misc.h>
#else
    #define WOLFSSL_MISC_INCLUDED
    #define WOLFSSL_HAVE_MIN
    #define WOLFSSL_HAVE_MAX
    #include <wolfcrypt/src/misc.c>
#endif

#if defined(STM32_CRYPTO)
#elif defined(HAVE_COLDFIRE_SEC)
#elif defined(FREESCALE_LTC)
#elif defined(FREESCALE_MMCAU)
#elif defined(WOLFSSL_PIC32MZ_CRYPT)
#elif defined(WOLFSSL_NRF51_AES)
#elif defined(WOLFSSL_ESP32_CRYPT) && \
     !defined(NO_WOLFSSL_ESP32_CRYPT_AES)
#elif defined(WOLFSSL_AESNI)
#elif (defined(WOLFSSL_IMX6_CAAM) && !defined(NO_IMX6_CAAM_AES) \
        && !defined(WOLFSSL_QNX_CAAM)) || \
      ((defined(WOLFSSL_AFALG) || defined(WOLFSSL_DEVCRYPTO_AES)) && \
        defined(HAVE_AESCCM))
#elif defined(WOLFSSL_AFALG)
    /* implemented in wolfcrypt/src/port/af_alg/afalg_aes.c */

#elif defined(WOLFSSL_DEVCRYPTO_AES)
    /* implemented in wolfcrypt/src/port/devcrypto/devcrypto_aes.c */

#elif defined(WOLFSSL_SCE) && !defined(WOLFSSL_SCE_NO_AES)
#elif defined(WOLFSSL_KCAPI_AES)
#elif defined(WOLFSSL_HAVE_PSA) && !defined(WOLFSSL_PSA_NO_AES)
/* implemented in wolfcrypt/src/port/psa/psa_aes.c */

#else

    /* using wolfCrypt software implementation */
    #define NEED_AES_TABLES
#endif

#if !defined(NO_AES) && !defined(WOLFSSL_TI_CRYPT) && !defined(WOLFSSL_ARMASM) && \
    defined(NEED_AES_TABLES) && (defined(HAVE_AES_CBC) || defined(WOLFSSL_AES_DIRECT) || defined(HAVE_AESCCM) || defined(HAVE_AESGCM)) && \
    defined(HAVE_CUDA)

#define GETBYTE(x, y) (word32)((byte)((x) >> (8 * (y))))

#ifndef WC_CACHE_LINE_SZ
    #if defined(__x86_64__) || defined(_M_X64) || \
       (defined(__ILP32__) && (__ILP32__ >= 1))
        #define WC_CACHE_LINE_SZ 64
    #else
        /* default cache line size */
        #define WC_CACHE_LINE_SZ 32
    #endif
#endif

#ifndef WOLFSSL_AES_SMALL_TABLES
extern const FLASH_QUALIFIER word32 Te[4][256];
__global__
static word32 GetTable(const word32* t, byte o, word32 *e)
{
#if WC_CACHE_LINE_SZ == 64
  byte hi = o & 0xf0;
  byte lo = o & 0x0f;

  *e  = t[lo + 0x00] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  *e |= t[lo + 0x10] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  *e |= t[lo + 0x20] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  *e |= t[lo + 0x30] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  *e |= t[lo + 0x40] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  *e |= t[lo + 0x50] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  *e |= t[lo + 0x60] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  *e |= t[lo + 0x70] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  *e |= t[lo + 0x80] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  *e |= t[lo + 0x90] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  *e |= t[lo + 0xa0] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  *e |= t[lo + 0xb0] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  *e |= t[lo + 0xc0] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  *e |= t[lo + 0xd0] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  *e |= t[lo + 0xe0] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  *e |= t[lo + 0xf0] & ((word32)0 - (((word32)hi - 0x01) >> 31));
#else
  *e = 0;
  int i;
  byte hi = o & WC_CACHE_LINE_MASK_HI;
  byte lo = o & WC_CACHE_LINE_MASK_LO;

  for (i = 0; i < 256; i += (1 << WC_CACHE_LINE_BITS)) {
      *e |= t[lo + i] & ((word32)0 - (((word32)hi - 0x01) >> 31));
      hi -= WC_CACHE_LINE_ADD;
  }
#endif
}

__global__
static void GetTable_Multi(const word32* t, word32* t0, byte o0,
  word32* t1, byte o1, word32* t2, byte o2, word32* t3, byte o3)
{
  word32 e0 = 0;
  word32 e1 = 0;
  word32 e2 = 0;
  word32 e3 = 0;
  byte hi0 = o0 & WC_CACHE_LINE_MASK_HI;
  byte lo0 = o0 & WC_CACHE_LINE_MASK_LO;
  byte hi1 = o1 & WC_CACHE_LINE_MASK_HI;
  byte lo1 = o1 & WC_CACHE_LINE_MASK_LO;
  byte hi2 = o2 & WC_CACHE_LINE_MASK_HI;
  byte lo2 = o2 & WC_CACHE_LINE_MASK_LO;
  byte hi3 = o3 & WC_CACHE_LINE_MASK_HI;
  byte lo3 = o3 & WC_CACHE_LINE_MASK_LO;
  int i;

  for (i = 0; i < 256; i += (1 << WC_CACHE_LINE_BITS)) {
      e0 |= t[lo0 + i] & ((word32)0 - (((word32)hi0 - 0x01) >> 31));
      hi0 -= WC_CACHE_LINE_ADD;
      e1 |= t[lo1 + i] & ((word32)0 - (((word32)hi1 - 0x01) >> 31));
      hi1 -= WC_CACHE_LINE_ADD;
      e2 |= t[lo2 + i] & ((word32)0 - (((word32)hi2 - 0x01) >> 31));
      hi2 -= WC_CACHE_LINE_ADD;
      e3 |= t[lo3 + i] & ((word32)0 - (((word32)hi3 - 0x01) >> 31));
      hi3 -= WC_CACHE_LINE_ADD;
  }
  *t0 = e0;
  *t1 = e1;
  *t2 = e2;
  *t3 = e3;
}

/* load 4 Te Tables into cache by cache line stride */
static WARN_UNUSED_RESULT WC_INLINE word32 PreFetchTe(void)
{
#ifndef WOLFSSL_AES_TOUCH_LINES
    word32 x = 0;
    int i,j;

    for (i = 0; i < 4; i++) {
        /* 256 elements, each one is 4 bytes */
        for (j = 0; j < 256; j += WC_CACHE_LINE_SZ/4) {
            x &= Te[i][j];
        }
    }
    return x;
#else
    return 0;
#endif
}
#else
extern __device__ const byte Tsbox[256];
#define AES_XTIME(x)    ((byte)((byte)((x) << 1) ^ ((0 - ((x) >> 7)) & 0x1b)))

#define col_mul(t, i2, i3, ia, ib) \
  ( GETBYTE(t, ia) ^ GETBYTE(t, ib) ^ GETBYTE(t, i3) ^ AES_XTIME(GETBYTE(t, i2) ^ GETBYTE(t, i3)) )

#define GetTable(t, o)  t[o]
#define GetTable8(t, o) t[o]
#define GetTable_Multi(t, t0, o0, t1, o1, t2, o2, t3, o3)  \
  *(t0) = (t)[o0]; *(t1) = (t)[o1]; *(t2) = (t)[o2]; *(t3) = (t)[o3]
#define XorTable_Multi(t, t0, o0, t1, o1, t2, o2, t3, o3)  \
  *(t0) ^= (t)[o0]; *(t1) ^= (t)[o1]; *(t2) ^= (t)[o2]; *(t3) ^= (t)[o3]
#define GetTable8_4(t, o0, o1, o2, o3) \
  (((word32)(t)[o0] << 24) | ((word32)(t)[o1] << 16) |   \
   ((word32)(t)[o2] <<  8) | ((word32)(t)[o3] <<  0))

/* load sbox into cache by cache line stride */
static WARN_UNUSED_RESULT WC_INLINE word32 PreFetchSBox(void)
{
#ifndef WOLFSSL_AES_TOUCH_LINES
    word32 x = 0;
    int i;

    for (i = 0; i < 256; i += WC_CACHE_LINE_SZ/4) {
        x &= Tsbox[i];
    }
    return x;
#else
    return 0;
#endif
}
#endif

#if !defined(WC_AES_BITSLICED)
/* Encrypt a block using AES.
 *
 * @param [in]  aes       AES object.
 * @param [in]  inBlock   Block to encrypt.
 * @param [out] outBlock  Encrypted block.
 * @param [in]  r         Rounds divided by 2.
 */
__global__ void AesEncrypt_C(Aes* aes, const byte* inBlock, byte* outBlock,
        word32 r)
{
    word32 s0, s1, s2, s3;
    word32 t0, t1, t2, t3;
    const word32* rk;

#ifdef WC_AES_C_DYNAMIC_FALLBACK
    rk = aes->key_C_fallback;
#else
    rk = aes->key;
#endif

    /*
     * map byte array block to cipher state
     * and add initial round key:
     */
    XMEMCPY(&s0, inBlock,                  sizeof(s0));
    XMEMCPY(&s1, inBlock +     sizeof(s0), sizeof(s1));
    XMEMCPY(&s2, inBlock + 2 * sizeof(s0), sizeof(s2));
    XMEMCPY(&s3, inBlock + 3 * sizeof(s0), sizeof(s3));

#ifdef LITTLE_ENDIAN_ORDER
    s0 = ByteReverseWord32(s0);
    s1 = ByteReverseWord32(s1);
    s2 = ByteReverseWord32(s2);
    s3 = ByteReverseWord32(s3);
#endif

    /* AddRoundKey */
    s0 ^= rk[0];
    s1 ^= rk[1];
    s2 ^= rk[2];
    s3 ^= rk[3];

#ifndef WOLFSSL_AES_SMALL_TABLES
#ifndef WC_NO_CACHE_RESISTANT
    s0 |= PreFetchTe();
#endif

#ifndef WOLFSSL_AES_TOUCH_LINES
#define ENC_ROUND_T_S(o)                                                       \
    t0 = GetTable<<<1,1>>>(Te[0], GETBYTE(s0, 3)) ^ GetTable<<<1,1>>>(Te[1], GETBYTE(s1, 2)) ^   \
         GetTable<<<1,1>>>(Te[2], GETBYTE(s2, 1)) ^ GetTable<<<1,1>>>(Te[3], GETBYTE(s3, 0)) ^   \
         rk[(o)+4];                                                            \
    t1 = GetTable<<<1,1>>>(Te[0], GETBYTE(s1, 3)) ^ GetTable<<<1,1>>>(Te[1], GETBYTE(s2, 2)) ^   \
         GetTable<<<1,1>>>(Te[2], GETBYTE(s3, 1)) ^ GetTable<<<1,1>>>(Te[3], GETBYTE(s0, 0)) ^   \
         rk[(o)+5];                                                            \
    t2 = GetTable<<<1,1>>>(Te[0], GETBYTE(s2, 3)) ^ GetTable<<<1,1>>>(Te[1], GETBYTE(s3, 2)) ^   \
         GetTable<<<1,1>>>(Te[2], GETBYTE(s0, 1)) ^ GetTable<<<1,1>>>(Te[3], GETBYTE(s1, 0)) ^   \
         rk[(o)+6];                                                            \
    t3 = GetTable<<<1,1>>>(Te[0], GETBYTE(s3, 3)) ^ GetTable<<<1,1>>>(Te[1], GETBYTE(s0, 2)) ^   \
         GetTable<<<1,1>>>(Te[2], GETBYTE(s1, 1)) ^ GetTable<<<1,1>>>(Te[3], GETBYTE(s2, 0)) ^   \
         rk[(o)+7]
#define ENC_ROUND_S_T(o)                                                       \
    s0 = GetTable<<<1,1>>>(Te[0], GETBYTE(t0, 3)) ^ GetTable<<<1,1>>>(Te[1], GETBYTE(t1, 2)) ^   \
         GetTable<<<1,1>>>(Te[2], GETBYTE(t2, 1)) ^ GetTable<<<1,1>>>(Te[3], GETBYTE(t3, 0)) ^   \
         rk[(o)+0];                                                            \
    s1 = GetTable<<<1,1>>>(Te[0], GETBYTE(t1, 3)) ^ GetTable<<<1,1>>>(Te[1], GETBYTE(t2, 2)) ^   \
         GetTable<<<1,1>>>(Te[2], GETBYTE(t3, 1)) ^ GetTable<<<1,1>>>(Te[3], GETBYTE(t0, 0)) ^   \
         rk[(o)+1];                                                            \
    s2 = GetTable<<<1,1>>>(Te[0], GETBYTE(t2, 3)) ^ GetTable<<<1,1>>>(Te[1], GETBYTE(t3, 2)) ^   \
         GetTable<<<1,1>>>(Te[2], GETBYTE(t0, 1)) ^ GetTable<<<1,1>>>(Te[3], GETBYTE(t1, 0)) ^   \
         rk[(o)+2];                                                            \
    s3 = GetTable<<<1,1>>>(Te[0], GETBYTE(t3, 3)) ^ GetTable<<<1,1>>>(Te[1], GETBYTE(t0, 2)) ^   \
         GetTable<<<1,1>>>(Te[2], GETBYTE(t1, 1)) ^ GetTable<<<1,1>>>(Te[3], GETBYTE(t2, 0)) ^   \
         rk[(o)+3]
#else
#define ENC_ROUND_T_S(o)                                                       \
    GetTable_Multi<<<1,1>>>(Te[0], &t0, GETBYTE(s0, 3), &t1, GETBYTE(s1, 3),            \
                          &t2, GETBYTE(s2, 3), &t3, GETBYTE(s3, 3));           \
    XorTable_Multi(Te[1], &t0, GETBYTE(s1, 2), &t1, GETBYTE(s2, 2),            \
                          &t2, GETBYTE(s3, 2), &t3, GETBYTE(s0, 2));           \
    XorTable_Multi(Te[2], &t0, GETBYTE(s2, 1), &t1, GETBYTE(s3, 1),            \
                          &t2, GETBYTE(s0, 1), &t3, GETBYTE(s1, 1));           \
    XorTable_Multi(Te[3], &t0, GETBYTE(s3, 0), &t1, GETBYTE(s0, 0),            \
                          &t2, GETBYTE(s1, 0), &t3, GETBYTE(s2, 0));           \
    t0 ^= rk[(o)+4]; t1 ^= rk[(o)+5]; t2 ^= rk[(o)+6]; t3 ^= rk[(o)+7];

#define ENC_ROUND_S_T(o)                                                       \
    GetTable_Multi<<<1,1>>>(Te[0], &s0, GETBYTE(t0, 3), &s1, GETBYTE(t1, 3),            \
                          &s2, GETBYTE(t2, 3), &s3, GETBYTE(t3, 3));           \
    XorTable_Multi(Te[1], &s0, GETBYTE(t1, 2), &s1, GETBYTE(t2, 2),            \
                          &s2, GETBYTE(t3, 2), &s3, GETBYTE(t0, 2));           \
    XorTable_Multi(Te[2], &s0, GETBYTE(t2, 1), &s1, GETBYTE(t3, 1),            \
                          &s2, GETBYTE(t0, 1), &s3, GETBYTE(t1, 1));           \
    XorTable_Multi(Te[3], &s0, GETBYTE(t3, 0), &s1, GETBYTE(t0, 0),            \
                          &s2, GETBYTE(t1, 0), &s3, GETBYTE(t2, 0));           \
    s0 ^= rk[(o)+0]; s1 ^= rk[(o)+1]; s2 ^= rk[(o)+2]; s3 ^= rk[(o)+3];
#endif

#ifndef WOLFSSL_AES_NO_UNROLL
/* Unroll the loop. */
                       ENC_ROUND_T_S( 0);
    ENC_ROUND_S_T( 8); ENC_ROUND_T_S( 8);
    ENC_ROUND_S_T(16); ENC_ROUND_T_S(16);
    ENC_ROUND_S_T(24); ENC_ROUND_T_S(24);
    ENC_ROUND_S_T(32); ENC_ROUND_T_S(32);
    if (r > 5) {
        ENC_ROUND_S_T(40); ENC_ROUND_T_S(40);
        if (r > 6) {
            ENC_ROUND_S_T(48); ENC_ROUND_T_S(48);
        }
    }
    rk += r * 8;
#else
    /*
     * Nr - 1 full rounds:
     */

    for (;;) {
        ENC_ROUND_T_S(0);

        rk += 8;
        if (--r == 0) {
            break;
        }

        ENC_ROUND_S_T(0);
    }
#endif

    /*
     * apply last round and
     * map cipher state to byte array block:
     */

#ifndef WOLFSSL_AES_TOUCH_LINES
    s0 =
        (GetTable(Te[2], GETBYTE(t0, 3)) & 0xff000000) ^
        (GetTable(Te[3], GETBYTE(t1, 2)) & 0x00ff0000) ^
        (GetTable(Te[0], GETBYTE(t2, 1)) & 0x0000ff00) ^
        (GetTable(Te[1], GETBYTE(t3, 0)) & 0x000000ff) ^
        rk[0];
    s1 =
        (GetTable(Te[2], GETBYTE(t1, 3)) & 0xff000000) ^
        (GetTable(Te[3], GETBYTE(t2, 2)) & 0x00ff0000) ^
        (GetTable(Te[0], GETBYTE(t3, 1)) & 0x0000ff00) ^
        (GetTable(Te[1], GETBYTE(t0, 0)) & 0x000000ff) ^
        rk[1];
    s2 =
        (GetTable(Te[2], GETBYTE(t2, 3)) & 0xff000000) ^
        (GetTable(Te[3], GETBYTE(t3, 2)) & 0x00ff0000) ^
        (GetTable(Te[0], GETBYTE(t0, 1)) & 0x0000ff00) ^
        (GetTable(Te[1], GETBYTE(t1, 0)) & 0x000000ff) ^
        rk[2];
    s3 =
        (GetTable(Te[2], GETBYTE(t3, 3)) & 0xff000000) ^
        (GetTable(Te[3], GETBYTE(t0, 2)) & 0x00ff0000) ^
        (GetTable(Te[0], GETBYTE(t1, 1)) & 0x0000ff00) ^
        (GetTable(Te[1], GETBYTE(t2, 0)) & 0x000000ff) ^
        rk[3];
#else
{
    word32 u0;
    word32 u1;
    word32 u2;
    word32 u3;

    s0 = rk[0]; s1 = rk[1]; s2 = rk[2]; s3 = rk[3];
    GetTable_Multi(Te[2], &u0, GETBYTE(t0, 3), &u1, GETBYTE(t1, 3),
                          &u2, GETBYTE(t2, 3), &u3, GETBYTE(t3, 3));
    s0 ^= u0 & 0xff000000; s1 ^= u1 & 0xff000000;
    s2 ^= u2 & 0xff000000; s3 ^= u3 & 0xff000000;
    GetTable_Multi(Te[3], &u0, GETBYTE(t1, 2), &u1, GETBYTE(t2, 2),
                          &u2, GETBYTE(t3, 2), &u3, GETBYTE(t0, 2));
    s0 ^= u0 & 0x00ff0000; s1 ^= u1 & 0x00ff0000;
    s2 ^= u2 & 0x00ff0000; s3 ^= u3 & 0x00ff0000;
    GetTable_Multi(Te[0], &u0, GETBYTE(t2, 1), &u1, GETBYTE(t3, 1),
                          &u2, GETBYTE(t0, 1), &u3, GETBYTE(t1, 1));
    s0 ^= u0 & 0x0000ff00; s1 ^= u1 & 0x0000ff00;
    s2 ^= u2 & 0x0000ff00; s3 ^= u3 & 0x0000ff00;
    GetTable_Multi(Te[1], &u0, GETBYTE(t3, 0), &u1, GETBYTE(t0, 0),
                          &u2, GETBYTE(t1, 0), &u3, GETBYTE(t2, 0));
    s0 ^= u0 & 0x000000ff; s1 ^= u1 & 0x000000ff;
    s2 ^= u2 & 0x000000ff; s3 ^= u3 & 0x000000ff;
}
#endif
#else
#ifndef WC_NO_CACHE_RESISTANT
    s0 |= PreFetchSBox();
#endif

    r *= 2;
    /* Two rounds at a time */
    for (rk += 4; r > 1; r--, rk += 4) {
        t0 =
            ((word32)GetTable8(Tsbox, GETBYTE(s0, 3)) << 24) ^
            ((word32)GetTable8(Tsbox, GETBYTE(s1, 2)) << 16) ^
            ((word32)GetTable8(Tsbox, GETBYTE(s2, 1)) <<  8) ^
            ((word32)GetTable8(Tsbox, GETBYTE(s3, 0)));
        t1 =
            ((word32)GetTable8(Tsbox, GETBYTE(s1, 3)) << 24) ^
            ((word32)GetTable8(Tsbox, GETBYTE(s2, 2)) << 16) ^
            ((word32)GetTable8(Tsbox, GETBYTE(s3, 1)) <<  8) ^
            ((word32)GetTable8(Tsbox, GETBYTE(s0, 0)));
        t2 =
            ((word32)GetTable8(Tsbox, GETBYTE(s2, 3)) << 24) ^
            ((word32)GetTable8(Tsbox, GETBYTE(s3, 2)) << 16) ^
            ((word32)GetTable8(Tsbox, GETBYTE(s0, 1)) <<  8) ^
            ((word32)GetTable8(Tsbox, GETBYTE(s1, 0)));
        t3 =
            ((word32)GetTable8(Tsbox, GETBYTE(s3, 3)) << 24) ^
            ((word32)GetTable8(Tsbox, GETBYTE(s0, 2)) << 16) ^
            ((word32)GetTable8(Tsbox, GETBYTE(s1, 1)) <<  8) ^
            ((word32)GetTable8(Tsbox, GETBYTE(s2, 0)));

        s0 =
            (col_mul(t0, 3, 2, 0, 1) << 24) ^
            (col_mul(t0, 2, 1, 0, 3) << 16) ^
            (col_mul(t0, 1, 0, 2, 3) <<  8) ^
            (col_mul(t0, 0, 3, 2, 1)      ) ^
            rk[0];
        s1 =
            (col_mul(t1, 3, 2, 0, 1) << 24) ^
            (col_mul(t1, 2, 1, 0, 3) << 16) ^
            (col_mul(t1, 1, 0, 2, 3) <<  8) ^
            (col_mul(t1, 0, 3, 2, 1)      ) ^
            rk[1];
        s2 =
            (col_mul(t2, 3, 2, 0, 1) << 24) ^
            (col_mul(t2, 2, 1, 0, 3) << 16) ^
            (col_mul(t2, 1, 0, 2, 3) <<  8) ^
            (col_mul(t2, 0, 3, 2, 1)      ) ^
            rk[2];
        s3 =
            (col_mul(t3, 3, 2, 0, 1) << 24) ^
            (col_mul(t3, 2, 1, 0, 3) << 16) ^
            (col_mul(t3, 1, 0, 2, 3) <<  8) ^
            (col_mul(t3, 0, 3, 2, 1)      ) ^
            rk[3];
    }

    t0 =
        ((word32)GetTable8(Tsbox, GETBYTE(s0, 3)) << 24) ^
        ((word32)GetTable8(Tsbox, GETBYTE(s1, 2)) << 16) ^
        ((word32)GetTable8(Tsbox, GETBYTE(s2, 1)) <<  8) ^
        ((word32)GetTable8(Tsbox, GETBYTE(s3, 0)));
    t1 =
        ((word32)GetTable8(Tsbox, GETBYTE(s1, 3)) << 24) ^
        ((word32)GetTable8(Tsbox, GETBYTE(s2, 2)) << 16) ^
        ((word32)GetTable8(Tsbox, GETBYTE(s3, 1)) <<  8) ^
        ((word32)GetTable8(Tsbox, GETBYTE(s0, 0)));
    t2 =
        ((word32)GetTable8(Tsbox, GETBYTE(s2, 3)) << 24) ^
        ((word32)GetTable8(Tsbox, GETBYTE(s3, 2)) << 16) ^
        ((word32)GetTable8(Tsbox, GETBYTE(s0, 1)) <<  8) ^
        ((word32)GetTable8(Tsbox, GETBYTE(s1, 0)));
    t3 =
        ((word32)GetTable8(Tsbox, GETBYTE(s3, 3)) << 24) ^
        ((word32)GetTable8(Tsbox, GETBYTE(s0, 2)) << 16) ^
        ((word32)GetTable8(Tsbox, GETBYTE(s1, 1)) <<  8) ^
        ((word32)GetTable8(Tsbox, GETBYTE(s2, 0)));
    s0 = t0 ^ rk[0];
    s1 = t1 ^ rk[1];
    s2 = t2 ^ rk[2];
    s3 = t3 ^ rk[3];
#endif

    /* write out */
#ifdef LITTLE_ENDIAN_ORDER
    s0 = ByteReverseWord32(s0);
    s1 = ByteReverseWord32(s1);
    s2 = ByteReverseWord32(s2);
    s3 = ByteReverseWord32(s3);
#endif

    XMEMCPY(outBlock,                  &s0, sizeof(s0));
    XMEMCPY(outBlock +     sizeof(s0), &s1, sizeof(s1));
    XMEMCPY(outBlock + 2 * sizeof(s0), &s2, sizeof(s2));
    XMEMCPY(outBlock + 3 * sizeof(s0), &s3, sizeof(s3));
}

#if defined(HAVE_AES_ECB) && !(defined(WOLFSSL_IMX6_CAAM) && \
    !defined(NO_IMX6_CAAM_AES) && !defined(WOLFSSL_QNX_CAAM))
/* Encrypt a number of blocks using AES.
 *
 * @param [in]  aes  AES object.
 * @param [in]  in   Block to encrypt.
 * @param [out] out  Encrypted block.
 * @param [in]  sz   Number of blocks to encrypt.
 */
void AesEncryptBlocks_C(Aes* aes, const byte* in, byte* out, word32 sz)
{
    word32 i;

    for (i = 0; i < sz; i += AES_BLOCK_SIZE) {
        AesEncrypt_C<<<1,1>>>(aes, in, out, aes->rounds >> 1);
        in += AES_BLOCK_SIZE;
        out += AES_BLOCK_SIZE;
    }
}
#endif

#else

/* Encrypt a block using AES.
 *
 * @param [in]  aes       AES object.
 * @param [in]  inBlock   Block to encrypt.
 * @param [out] outBlock  Encrypted block.
 * @param [in]  r         Rounds divided by 2.
 */
__global__
void AesEncrypt_C(Aes* aes, const byte* inBlock, byte* outBlock,
        word32 r)
{
    bs_word state[AES_BLOCK_BITS];

    (void)r;

    XMEMCPY(state, inBlock, AES_BLOCK_SIZE);
    XMEMSET(((byte*)state) + AES_BLOCK_SIZE, 0, sizeof(state) - AES_BLOCK_SIZE);

    bs_encrypt(state, aes->bs_key, aes->rounds);

    XMEMCPY(outBlock, state, AES_BLOCK_SIZE);
}

#if defined(HAVE_AES_ECB) && !(defined(WOLFSSL_IMX6_CAAM) && \
    !defined(NO_IMX6_CAAM_AES) && !defined(WOLFSSL_QNX_CAAM))
/* Encrypt a number of blocks using AES.
 *
 * @param [in]  aes  AES object.
 * @param [in]  in   Block to encrypt.
 * @param [out] out  Encrypted block.
 * @param [in]  sz   Number of blocks to encrypt.
 */
void AesEncryptBlocks_C(Aes* aes, const byte* in, byte* out, word32 sz)
{
    bs_word state[AES_BLOCK_BITS];

    while (sz >= BS_BLOCK_SIZE) {
        XMEMCPY(state, in, BS_BLOCK_SIZE);
        bs_encrypt(state, aes->bs_key, aes->rounds);
        XMEMCPY(out, state, BS_BLOCK_SIZE);
        sz  -= BS_BLOCK_SIZE;
        in  += BS_BLOCK_SIZE;
        out += BS_BLOCK_SIZE;
    }
    if (sz > 0) {
        XMEMCPY(state, in, sz);
        XMEMSET(((byte*)state) + sz, 0, sizeof(state) - sz);
        bs_encrypt(state, aes->bs_key, aes->rounds);
        XMEMCPY(out, state, sz);
    }
}
#endif

#endif /* !WC_AES_BITSLICED */

#endif /* HAVE_CUDA */

#endif /* !WOLFSSL_TI_CRYPT */

