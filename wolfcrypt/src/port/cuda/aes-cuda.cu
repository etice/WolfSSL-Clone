#include "hip/hip_runtime.h"
/* aes.c
 *
 * Copyright (C) 2006-2023 wolfSSL Inc.
 *
 * This file is part of wolfSSL.
 *
 * wolfSSL is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License, or
 * (at your option) any later version.
 *
 * wolfSSL is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA 02110-1335, USA
 */

/*

DESCRIPTION
This library provides the interfaces to the Advanced Encryption Standard (AES)
for encrypting and decrypting data. AES is the standard known for a symmetric
block cipher mechanism that uses n-bit binary string parameter key with 128-bits,
192-bits, and 256-bits of key sizes.

*/
#ifdef HAVE_CONFIG_H
    #include <config.h>
#endif

#include <wolfssl/wolfcrypt/settings.h>
#include <wolfssl/wolfcrypt/error-crypt.h>

#include <wolfssl/wolfcrypt/aes.h>

#ifdef WOLFSSL_AESNI
#include <wmmintrin.h>
#include <emmintrin.h>
#include <smmintrin.h>
#endif /* WOLFSSL_AESNI */

#include <wolfssl/wolfcrypt/cpuid.h>

#ifdef WOLF_CRYPTO_CB
    #include <wolfssl/wolfcrypt/cryptocb.h>
#endif

#ifdef WOLFSSL_SECO_CAAM
#include <wolfssl/wolfcrypt/port/caam/wolfcaam.h>
#endif

#ifdef WOLFSSL_IMXRT_DCP
    #include <wolfssl/wolfcrypt/port/nxp/dcp_port.h>
#endif
#if defined(WOLFSSL_SE050) && defined(WOLFSSL_SE050_CRYPT)
    #include <wolfssl/wolfcrypt/port/nxp/se050_port.h>
#endif

#if defined(WOLFSSL_AES_SIV)
    #include <wolfssl/wolfcrypt/cmac.h>
#endif /* WOLFSSL_AES_SIV */

#if defined(WOLFSSL_HAVE_PSA) && !defined(WOLFSSL_PSA_NO_AES)
    #include <wolfssl/wolfcrypt/port/psa/psa.h>
#endif

#if defined(WOLFSSL_TI_CRYPT)
    #include <wolfcrypt/src/port/ti/ti-aes.c>
#else

#include <wolfssl/wolfcrypt/logging.h>

#ifdef NO_INLINE
    #include <wolfssl/wolfcrypt/misc.h>
#else
    #define WOLFSSL_MISC_INCLUDED
    #define WOLFSSL_HAVE_MIN
    #define WOLFSSL_HAVE_MAX
//    #include <wolfcrypt/src/misc.c>
#endif
/* This routine performs a left circular arithmetic shift of <x> by <y> value. */

#define rotlFixed(x, y) ( (x << y) | (x >> (sizeof(x) * 8 - y)) )

/* This routine performs a right circular arithmetic shift of <x> by <y> value. */
#define rotrFixed(x, y) ( (x >> y) | (x << (sizeof(x) * 8 - y)) )

#ifdef WC_RC2

/* This routine performs a left circular arithmetic shift of <x> by <y> value */
WC_MISC_STATIC WC_INLINE word16 rotlFixed16(word16 x, word16 y)
{
    return (x << y) | (x >> (sizeof(x) * 8 - y));
}


/* This routine performs a right circular arithmetic shift of <x> by <y> value */
WC_MISC_STATIC WC_INLINE word16 rotrFixed16(word16 x, word16 y)
{
    return (x >> y) | (x << (sizeof(x) * 8 - y));
}

#endif /* WC_RC2 */

/* This routine performs a byte swap of 32-bit word value. */
#if defined(__CCRX__) && !defined(NO_INLINE) /* shortest version for CC-RX */
    #define ByteReverseWord32(value, outRef) ( *outRef = _builtin_revl(value) )
#else
    #define ByteReverseWord32(value, outRef) ( *outRef = rotlFixed( ((value & 0xFF00FF00) >> 8) | ((value & 0x00FF00FF) << 8) , 16U) )
#endif /* ! (__CCRX__ && !NO_INLINE) */

#if defined(STM32_CRYPTO)
#elif defined(HAVE_COLDFIRE_SEC)
#elif defined(FREESCALE_LTC)
#elif defined(FREESCALE_MMCAU)
#elif defined(WOLFSSL_PIC32MZ_CRYPT)
#elif defined(WOLFSSL_NRF51_AES)
#elif defined(WOLFSSL_ESP32_CRYPT) && \
     !defined(NO_WOLFSSL_ESP32_CRYPT_AES)
#elif defined(WOLFSSL_AESNI)
#elif (defined(WOLFSSL_IMX6_CAAM) && !defined(NO_IMX6_CAAM_AES) \
        && !defined(WOLFSSL_QNX_CAAM)) || \
      ((defined(WOLFSSL_AFALG) || defined(WOLFSSL_DEVCRYPTO_AES)) && \
        defined(HAVE_AESCCM))
#elif defined(WOLFSSL_AFALG)
    /* implemented in wolfcrypt/src/port/af_alg/afalg_aes.c */

#elif defined(WOLFSSL_DEVCRYPTO_AES)
    /* implemented in wolfcrypt/src/port/devcrypto/devcrypto_aes.c */

#elif defined(WOLFSSL_SCE) && !defined(WOLFSSL_SCE_NO_AES)
#elif defined(WOLFSSL_KCAPI_AES)
#elif defined(WOLFSSL_HAVE_PSA) && !defined(WOLFSSL_PSA_NO_AES)
/* implemented in wolfcrypt/src/port/psa/psa_aes.c */

#else

    /* using wolfCrypt software implementation */
    #define NEED_AES_TABLES
#endif

#if !defined(NO_AES) && !defined(WOLFSSL_TI_CRYPT) && !defined(WOLFSSL_ARMASM) && \
    defined(NEED_AES_TABLES) && (defined(HAVE_AES_CBC) || defined(WOLFSSL_AES_DIRECT) || defined(HAVE_AESCCM) || defined(HAVE_AESGCM)) && \
    defined(HAVE_CUDA)

#define GETBYTE(x, y) (word32)((byte)((x) >> (8 * (y))))

#ifndef WC_CACHE_LINE_SZ
    #if defined(__x86_64__) || defined(_M_X64) || \
       (defined(__ILP32__) && (__ILP32__ >= 1))
        #define WC_CACHE_LINE_SZ 64
    #else
        /* default cache line size */
        #define WC_CACHE_LINE_SZ 32
    #endif
#endif

#ifndef WOLFSSL_AES_SMALL_TABLES
extern const FLASH_QUALIFIER word32 Te[4][256];
__global__
static word32 GetTable(const word32* t, byte o, word32 *e)
{
#if WC_CACHE_LINE_SZ == 64
  byte hi = o & 0xf0;
  byte lo = o & 0x0f;

  *e  = t[lo + 0x00] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  *e |= t[lo + 0x10] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  *e |= t[lo + 0x20] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  *e |= t[lo + 0x30] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  *e |= t[lo + 0x40] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  *e |= t[lo + 0x50] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  *e |= t[lo + 0x60] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  *e |= t[lo + 0x70] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  *e |= t[lo + 0x80] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  *e |= t[lo + 0x90] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  *e |= t[lo + 0xa0] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  *e |= t[lo + 0xb0] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  *e |= t[lo + 0xc0] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  *e |= t[lo + 0xd0] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  *e |= t[lo + 0xe0] & ((word32)0 - (((word32)hi - 0x01) >> 31)); hi -= 0x10;
  *e |= t[lo + 0xf0] & ((word32)0 - (((word32)hi - 0x01) >> 31));
#else
  *e = 0;
  int i;
  byte hi = o & WC_CACHE_LINE_MASK_HI;
  byte lo = o & WC_CACHE_LINE_MASK_LO;

  for (i = 0; i < 256; i += (1 << WC_CACHE_LINE_BITS)) {
      *e |= t[lo + i] & ((word32)0 - (((word32)hi - 0x01) >> 31));
      hi -= WC_CACHE_LINE_ADD;
  }
#endif
}

__global__
static void GetTable_Multi(const word32* t, word32* t0, byte o0,
  word32* t1, byte o1, word32* t2, byte o2, word32* t3, byte o3)
{
  word32 e0 = 0;
  word32 e1 = 0;
  word32 e2 = 0;
  word32 e3 = 0;
  byte hi0 = o0 & WC_CACHE_LINE_MASK_HI;
  byte lo0 = o0 & WC_CACHE_LINE_MASK_LO;
  byte hi1 = o1 & WC_CACHE_LINE_MASK_HI;
  byte lo1 = o1 & WC_CACHE_LINE_MASK_LO;
  byte hi2 = o2 & WC_CACHE_LINE_MASK_HI;
  byte lo2 = o2 & WC_CACHE_LINE_MASK_LO;
  byte hi3 = o3 & WC_CACHE_LINE_MASK_HI;
  byte lo3 = o3 & WC_CACHE_LINE_MASK_LO;
  int i;

  for (i = 0; i < 256; i += (1 << WC_CACHE_LINE_BITS)) {
      e0 |= t[lo0 + i] & ((word32)0 - (((word32)hi0 - 0x01) >> 31));
      hi0 -= WC_CACHE_LINE_ADD;
      e1 |= t[lo1 + i] & ((word32)0 - (((word32)hi1 - 0x01) >> 31));
      hi1 -= WC_CACHE_LINE_ADD;
      e2 |= t[lo2 + i] & ((word32)0 - (((word32)hi2 - 0x01) >> 31));
      hi2 -= WC_CACHE_LINE_ADD;
      e3 |= t[lo3 + i] & ((word32)0 - (((word32)hi3 - 0x01) >> 31));
      hi3 -= WC_CACHE_LINE_ADD;
  }
  *t0 = e0;
  *t1 = e1;
  *t2 = e2;
  *t3 = e3;
}

/* load 4 Te Tables into cache by cache line stride */
static WARN_UNUSED_RESULT WC_INLINE word32 PreFetchTe(void)
{
#ifndef WOLFSSL_AES_TOUCH_LINES
    word32 x = 0;
    int i,j;

    for (i = 0; i < 4; i++) {
        /* 256 elements, each one is 4 bytes */
        for (j = 0; j < 256; j += WC_CACHE_LINE_SZ/4) {
            x &= Te[i][j];
        }
    }
    return x;
#else
    return 0;
#endif
}
#else
__device__ static const byte Tsbox[256] = {
    0x63U, 0x7cU, 0x77U, 0x7bU, 0xf2U, 0x6bU, 0x6fU, 0xc5U,
    0x30U, 0x01U, 0x67U, 0x2bU, 0xfeU, 0xd7U, 0xabU, 0x76U,
    0xcaU, 0x82U, 0xc9U, 0x7dU, 0xfaU, 0x59U, 0x47U, 0xf0U,
    0xadU, 0xd4U, 0xa2U, 0xafU, 0x9cU, 0xa4U, 0x72U, 0xc0U,
    0xb7U, 0xfdU, 0x93U, 0x26U, 0x36U, 0x3fU, 0xf7U, 0xccU,
    0x34U, 0xa5U, 0xe5U, 0xf1U, 0x71U, 0xd8U, 0x31U, 0x15U,
    0x04U, 0xc7U, 0x23U, 0xc3U, 0x18U, 0x96U, 0x05U, 0x9aU,
    0x07U, 0x12U, 0x80U, 0xe2U, 0xebU, 0x27U, 0xb2U, 0x75U,
    0x09U, 0x83U, 0x2cU, 0x1aU, 0x1bU, 0x6eU, 0x5aU, 0xa0U,
    0x52U, 0x3bU, 0xd6U, 0xb3U, 0x29U, 0xe3U, 0x2fU, 0x84U,
    0x53U, 0xd1U, 0x00U, 0xedU, 0x20U, 0xfcU, 0xb1U, 0x5bU,
    0x6aU, 0xcbU, 0xbeU, 0x39U, 0x4aU, 0x4cU, 0x58U, 0xcfU,
    0xd0U, 0xefU, 0xaaU, 0xfbU, 0x43U, 0x4dU, 0x33U, 0x85U,
    0x45U, 0xf9U, 0x02U, 0x7fU, 0x50U, 0x3cU, 0x9fU, 0xa8U,
    0x51U, 0xa3U, 0x40U, 0x8fU, 0x92U, 0x9dU, 0x38U, 0xf5U,
    0xbcU, 0xb6U, 0xdaU, 0x21U, 0x10U, 0xffU, 0xf3U, 0xd2U,
    0xcdU, 0x0cU, 0x13U, 0xecU, 0x5fU, 0x97U, 0x44U, 0x17U,
    0xc4U, 0xa7U, 0x7eU, 0x3dU, 0x64U, 0x5dU, 0x19U, 0x73U,
    0x60U, 0x81U, 0x4fU, 0xdcU, 0x22U, 0x2aU, 0x90U, 0x88U,
    0x46U, 0xeeU, 0xb8U, 0x14U, 0xdeU, 0x5eU, 0x0bU, 0xdbU,
    0xe0U, 0x32U, 0x3aU, 0x0aU, 0x49U, 0x06U, 0x24U, 0x5cU,
    0xc2U, 0xd3U, 0xacU, 0x62U, 0x91U, 0x95U, 0xe4U, 0x79U,
    0xe7U, 0xc8U, 0x37U, 0x6dU, 0x8dU, 0xd5U, 0x4eU, 0xa9U,
    0x6cU, 0x56U, 0xf4U, 0xeaU, 0x65U, 0x7aU, 0xaeU, 0x08U,
    0xbaU, 0x78U, 0x25U, 0x2eU, 0x1cU, 0xa6U, 0xb4U, 0xc6U,
    0xe8U, 0xddU, 0x74U, 0x1fU, 0x4bU, 0xbdU, 0x8bU, 0x8aU,
    0x70U, 0x3eU, 0xb5U, 0x66U, 0x48U, 0x03U, 0xf6U, 0x0eU,
    0x61U, 0x35U, 0x57U, 0xb9U, 0x86U, 0xc1U, 0x1dU, 0x9eU,
    0xe1U, 0xf8U, 0x98U, 0x11U, 0x69U, 0xd9U, 0x8eU, 0x94U,
    0x9bU, 0x1eU, 0x87U, 0xe9U, 0xceU, 0x55U, 0x28U, 0xdfU,
    0x8cU, 0xa1U, 0x89U, 0x0dU, 0xbfU, 0xe6U, 0x42U, 0x68U,
    0x41U, 0x99U, 0x2dU, 0x0fU, 0xb0U, 0x54U, 0xbbU, 0x16U
};

#define AES_XTIME(x)    ((byte)((byte)((x) << 1) ^ ((0 - ((x) >> 7)) & 0x1b)))

#define col_mul(t, i2, i3, ia, ib) \
  ( GETBYTE(t, ia) ^ GETBYTE(t, ib) ^ GETBYTE(t, i3) ^ AES_XTIME(GETBYTE(t, i2) ^ GETBYTE(t, i3)) )

#define GetTable(t, o)  t[o]
#define GetTable8(t, o) t[o]
#define GetTable_Multi(t, t0, o0, t1, o1, t2, o2, t3, o3)  \
  *(t0) = (t)[o0]; *(t1) = (t)[o1]; *(t2) = (t)[o2]; *(t3) = (t)[o3]
#define XorTable_Multi(t, t0, o0, t1, o1, t2, o2, t3, o3)  \
  *(t0) ^= (t)[o0]; *(t1) ^= (t)[o1]; *(t2) ^= (t)[o2]; *(t3) ^= (t)[o3]
#define GetTable8_4(t, o0, o1, o2, o3) \
  (((word32)(t)[o0] << 24) | ((word32)(t)[o1] << 16) |   \
   ((word32)(t)[o2] <<  8) | ((word32)(t)[o3] <<  0))

/* load sbox into cache by cache line stride */
#ifndef WOLFSSL_AES_TOUCH_LINES
    #define PreFetchSBox(x) { \
    x = 0; \
    int i; \
    for (i = 0; i < 256; i += WC_CACHE_LINE_SZ/4) { \
        x &= Tsbox[i]; \
    } \
    }
#else
    #define PreFetchSBox(x) ( x = 0 )
#endif
#endif

#if !defined(WC_AES_BITSLICED)
/* Encrypt a block using AES.
 *
 * @param [in]  aes       AES object.
 * @param [in]  inBlock   Block to encrypt.
 * @param [out] outBlock  Encrypted block.
 * @param [in]  r         Rounds divided by 2.
 */
__global__ void AesEncrypt_C_CUDA(Aes* aes, const byte* inBlock, byte* outBlock,
        word32 r)
{
    word32 s0, s1, s2, s3;
    word32 t0, t1, t2, t3;
    word32 sBox;
    const word32* rk;

#ifdef WC_AES_C_DYNAMIC_FALLBACK
    rk = aes->key_C_fallback;
#else
    rk = aes->key;
#endif

    /*
     * map byte array block to cipher state
     * and add initial round key:
     */
    XMEMCPY(&s0, inBlock,                  sizeof(s0));
    XMEMCPY(&s1, inBlock +     sizeof(s0), sizeof(s1));
    XMEMCPY(&s2, inBlock + 2 * sizeof(s0), sizeof(s2));
    XMEMCPY(&s3, inBlock + 3 * sizeof(s0), sizeof(s3));

#ifdef LITTLE_ENDIAN_ORDER
    ByteReverseWord32(s0,&s0);
    ByteReverseWord32(s1,&s1);
    ByteReverseWord32(s2,&s2);
    ByteReverseWord32(s3,&s3);
#endif

    /* AddRoundKey */
    s0 ^= rk[0];
    s1 ^= rk[1];
    s2 ^= rk[2];
    s3 ^= rk[3];

#ifndef WOLFSSL_AES_SMALL_TABLES
#ifndef WC_NO_CACHE_RESISTANT
    s0 |= PreFetchTe();
#endif

#ifndef WOLFSSL_AES_TOUCH_LINES
#define ENC_ROUND_T_S(o)                                                       \
    t0 = GetTable<<<1,1>>>(Te[0], GETBYTE(s0, 3)) ^ GetTable<<<1,1>>>(Te[1], GETBYTE(s1, 2)) ^   \
         GetTable<<<1,1>>>(Te[2], GETBYTE(s2, 1)) ^ GetTable<<<1,1>>>(Te[3], GETBYTE(s3, 0)) ^   \
         rk[(o)+4];                                                            \
    t1 = GetTable<<<1,1>>>(Te[0], GETBYTE(s1, 3)) ^ GetTable<<<1,1>>>(Te[1], GETBYTE(s2, 2)) ^   \
         GetTable<<<1,1>>>(Te[2], GETBYTE(s3, 1)) ^ GetTable<<<1,1>>>(Te[3], GETBYTE(s0, 0)) ^   \
         rk[(o)+5];                                                            \
    t2 = GetTable<<<1,1>>>(Te[0], GETBYTE(s2, 3)) ^ GetTable<<<1,1>>>(Te[1], GETBYTE(s3, 2)) ^   \
         GetTable<<<1,1>>>(Te[2], GETBYTE(s0, 1)) ^ GetTable<<<1,1>>>(Te[3], GETBYTE(s1, 0)) ^   \
         rk[(o)+6];                                                            \
    t3 = GetTable<<<1,1>>>(Te[0], GETBYTE(s3, 3)) ^ GetTable<<<1,1>>>(Te[1], GETBYTE(s0, 2)) ^   \
         GetTable<<<1,1>>>(Te[2], GETBYTE(s1, 1)) ^ GetTable<<<1,1>>>(Te[3], GETBYTE(s2, 0)) ^   \
         rk[(o)+7]
#define ENC_ROUND_S_T(o)                                                       \
    s0 = GetTable<<<1,1>>>(Te[0], GETBYTE(t0, 3)) ^ GetTable<<<1,1>>>(Te[1], GETBYTE(t1, 2)) ^   \
         GetTable<<<1,1>>>(Te[2], GETBYTE(t2, 1)) ^ GetTable<<<1,1>>>(Te[3], GETBYTE(t3, 0)) ^   \
         rk[(o)+0];                                                            \
    s1 = GetTable<<<1,1>>>(Te[0], GETBYTE(t1, 3)) ^ GetTable<<<1,1>>>(Te[1], GETBYTE(t2, 2)) ^   \
         GetTable<<<1,1>>>(Te[2], GETBYTE(t3, 1)) ^ GetTable<<<1,1>>>(Te[3], GETBYTE(t0, 0)) ^   \
         rk[(o)+1];                                                            \
    s2 = GetTable<<<1,1>>>(Te[0], GETBYTE(t2, 3)) ^ GetTable<<<1,1>>>(Te[1], GETBYTE(t3, 2)) ^   \
         GetTable<<<1,1>>>(Te[2], GETBYTE(t0, 1)) ^ GetTable<<<1,1>>>(Te[3], GETBYTE(t1, 0)) ^   \
         rk[(o)+2];                                                            \
    s3 = GetTable<<<1,1>>>(Te[0], GETBYTE(t3, 3)) ^ GetTable<<<1,1>>>(Te[1], GETBYTE(t0, 2)) ^   \
         GetTable<<<1,1>>>(Te[2], GETBYTE(t1, 1)) ^ GetTable<<<1,1>>>(Te[3], GETBYTE(t2, 0)) ^   \
         rk[(o)+3]
#else
#define ENC_ROUND_T_S(o)                                                       \
    GetTable_Multi<<<1,1>>>(Te[0], &t0, GETBYTE(s0, 3), &t1, GETBYTE(s1, 3),            \
                          &t2, GETBYTE(s2, 3), &t3, GETBYTE(s3, 3));           \
    XorTable_Multi(Te[1], &t0, GETBYTE(s1, 2), &t1, GETBYTE(s2, 2),            \
                          &t2, GETBYTE(s3, 2), &t3, GETBYTE(s0, 2));           \
    XorTable_Multi(Te[2], &t0, GETBYTE(s2, 1), &t1, GETBYTE(s3, 1),            \
                          &t2, GETBYTE(s0, 1), &t3, GETBYTE(s1, 1));           \
    XorTable_Multi(Te[3], &t0, GETBYTE(s3, 0), &t1, GETBYTE(s0, 0),            \
                          &t2, GETBYTE(s1, 0), &t3, GETBYTE(s2, 0));           \
    t0 ^= rk[(o)+4]; t1 ^= rk[(o)+5]; t2 ^= rk[(o)+6]; t3 ^= rk[(o)+7];

#define ENC_ROUND_S_T(o)                                                       \
    GetTable_Multi<<<1,1>>>(Te[0], &s0, GETBYTE(t0, 3), &s1, GETBYTE(t1, 3),            \
                          &s2, GETBYTE(t2, 3), &s3, GETBYTE(t3, 3));           \
    XorTable_Multi(Te[1], &s0, GETBYTE(t1, 2), &s1, GETBYTE(t2, 2),            \
                          &s2, GETBYTE(t3, 2), &s3, GETBYTE(t0, 2));           \
    XorTable_Multi(Te[2], &s0, GETBYTE(t2, 1), &s1, GETBYTE(t3, 1),            \
                          &s2, GETBYTE(t0, 1), &s3, GETBYTE(t1, 1));           \
    XorTable_Multi(Te[3], &s0, GETBYTE(t3, 0), &s1, GETBYTE(t0, 0),            \
                          &s2, GETBYTE(t1, 0), &s3, GETBYTE(t2, 0));           \
    s0 ^= rk[(o)+0]; s1 ^= rk[(o)+1]; s2 ^= rk[(o)+2]; s3 ^= rk[(o)+3];
#endif

#ifndef WOLFSSL_AES_NO_UNROLL
/* Unroll the loop. */
                       ENC_ROUND_T_S( 0);
    ENC_ROUND_S_T( 8); ENC_ROUND_T_S( 8);
    ENC_ROUND_S_T(16); ENC_ROUND_T_S(16);
    ENC_ROUND_S_T(24); ENC_ROUND_T_S(24);
    ENC_ROUND_S_T(32); ENC_ROUND_T_S(32);
    if (r > 5) {
        ENC_ROUND_S_T(40); ENC_ROUND_T_S(40);
        if (r > 6) {
            ENC_ROUND_S_T(48); ENC_ROUND_T_S(48);
        }
    }
    rk += r * 8;
#else
    /*
     * Nr - 1 full rounds:
     */

    for (;;) {
        ENC_ROUND_T_S(0);

        rk += 8;
        if (--r == 0) {
            break;
        }

        ENC_ROUND_S_T(0);
    }
#endif

    /*
     * apply last round and
     * map cipher state to byte array block:
     */

#ifndef WOLFSSL_AES_TOUCH_LINES
    s0 =
        (GetTable(Te[2], GETBYTE(t0, 3)) & 0xff000000) ^
        (GetTable(Te[3], GETBYTE(t1, 2)) & 0x00ff0000) ^
        (GetTable(Te[0], GETBYTE(t2, 1)) & 0x0000ff00) ^
        (GetTable(Te[1], GETBYTE(t3, 0)) & 0x000000ff) ^
        rk[0];
    s1 =
        (GetTable(Te[2], GETBYTE(t1, 3)) & 0xff000000) ^
        (GetTable(Te[3], GETBYTE(t2, 2)) & 0x00ff0000) ^
        (GetTable(Te[0], GETBYTE(t3, 1)) & 0x0000ff00) ^
        (GetTable(Te[1], GETBYTE(t0, 0)) & 0x000000ff) ^
        rk[1];
    s2 =
        (GetTable(Te[2], GETBYTE(t2, 3)) & 0xff000000) ^
        (GetTable(Te[3], GETBYTE(t3, 2)) & 0x00ff0000) ^
        (GetTable(Te[0], GETBYTE(t0, 1)) & 0x0000ff00) ^
        (GetTable(Te[1], GETBYTE(t1, 0)) & 0x000000ff) ^
        rk[2];
    s3 =
        (GetTable(Te[2], GETBYTE(t3, 3)) & 0xff000000) ^
        (GetTable(Te[3], GETBYTE(t0, 2)) & 0x00ff0000) ^
        (GetTable(Te[0], GETBYTE(t1, 1)) & 0x0000ff00) ^
        (GetTable(Te[1], GETBYTE(t2, 0)) & 0x000000ff) ^
        rk[3];
#else
{
    word32 u0;
    word32 u1;
    word32 u2;
    word32 u3;

    s0 = rk[0]; s1 = rk[1]; s2 = rk[2]; s3 = rk[3];
    GetTable_Multi(Te[2], &u0, GETBYTE(t0, 3), &u1, GETBYTE(t1, 3),
                          &u2, GETBYTE(t2, 3), &u3, GETBYTE(t3, 3));
    s0 ^= u0 & 0xff000000; s1 ^= u1 & 0xff000000;
    s2 ^= u2 & 0xff000000; s3 ^= u3 & 0xff000000;
    GetTable_Multi(Te[3], &u0, GETBYTE(t1, 2), &u1, GETBYTE(t2, 2),
                          &u2, GETBYTE(t3, 2), &u3, GETBYTE(t0, 2));
    s0 ^= u0 & 0x00ff0000; s1 ^= u1 & 0x00ff0000;
    s2 ^= u2 & 0x00ff0000; s3 ^= u3 & 0x00ff0000;
    GetTable_Multi(Te[0], &u0, GETBYTE(t2, 1), &u1, GETBYTE(t3, 1),
                          &u2, GETBYTE(t0, 1), &u3, GETBYTE(t1, 1));
    s0 ^= u0 & 0x0000ff00; s1 ^= u1 & 0x0000ff00;
    s2 ^= u2 & 0x0000ff00; s3 ^= u3 & 0x0000ff00;
    GetTable_Multi(Te[1], &u0, GETBYTE(t3, 0), &u1, GETBYTE(t0, 0),
                          &u2, GETBYTE(t1, 0), &u3, GETBYTE(t2, 0));
    s0 ^= u0 & 0x000000ff; s1 ^= u1 & 0x000000ff;
    s2 ^= u2 & 0x000000ff; s3 ^= u3 & 0x000000ff;
}
#endif
#else
#ifndef WC_NO_CACHE_RESISTANT
    PreFetchSBox(sBox);
    s0 |= sBox;
#endif

    r *= 2;
    /* Two rounds at a time */
    for (rk += 4; r > 1; r--, rk += 4) {
        t0 =
            ((word32)GetTable8(Tsbox, GETBYTE(s0, 3)) << 24) ^
            ((word32)GetTable8(Tsbox, GETBYTE(s1, 2)) << 16) ^
            ((word32)GetTable8(Tsbox, GETBYTE(s2, 1)) <<  8) ^
            ((word32)GetTable8(Tsbox, GETBYTE(s3, 0)));
        t1 =
            ((word32)GetTable8(Tsbox, GETBYTE(s1, 3)) << 24) ^
            ((word32)GetTable8(Tsbox, GETBYTE(s2, 2)) << 16) ^
            ((word32)GetTable8(Tsbox, GETBYTE(s3, 1)) <<  8) ^
            ((word32)GetTable8(Tsbox, GETBYTE(s0, 0)));
        t2 =
            ((word32)GetTable8(Tsbox, GETBYTE(s2, 3)) << 24) ^
            ((word32)GetTable8(Tsbox, GETBYTE(s3, 2)) << 16) ^
            ((word32)GetTable8(Tsbox, GETBYTE(s0, 1)) <<  8) ^
            ((word32)GetTable8(Tsbox, GETBYTE(s1, 0)));
        t3 =
            ((word32)GetTable8(Tsbox, GETBYTE(s3, 3)) << 24) ^
            ((word32)GetTable8(Tsbox, GETBYTE(s0, 2)) << 16) ^
            ((word32)GetTable8(Tsbox, GETBYTE(s1, 1)) <<  8) ^
            ((word32)GetTable8(Tsbox, GETBYTE(s2, 0)));

        s0 =
            (col_mul(t0, 3, 2, 0, 1) << 24) ^
            (col_mul(t0, 2, 1, 0, 3) << 16) ^
            (col_mul(t0, 1, 0, 2, 3) <<  8) ^
            (col_mul(t0, 0, 3, 2, 1)      ) ^
            rk[0];
        s1 =
            (col_mul(t1, 3, 2, 0, 1) << 24) ^
            (col_mul(t1, 2, 1, 0, 3) << 16) ^
            (col_mul(t1, 1, 0, 2, 3) <<  8) ^
            (col_mul(t1, 0, 3, 2, 1)      ) ^
            rk[1];
        s2 =
            (col_mul(t2, 3, 2, 0, 1) << 24) ^
            (col_mul(t2, 2, 1, 0, 3) << 16) ^
            (col_mul(t2, 1, 0, 2, 3) <<  8) ^
            (col_mul(t2, 0, 3, 2, 1)      ) ^
            rk[2];
        s3 =
            (col_mul(t3, 3, 2, 0, 1) << 24) ^
            (col_mul(t3, 2, 1, 0, 3) << 16) ^
            (col_mul(t3, 1, 0, 2, 3) <<  8) ^
            (col_mul(t3, 0, 3, 2, 1)      ) ^
            rk[3];
    }

    t0 =
        ((word32)GetTable8(Tsbox, GETBYTE(s0, 3)) << 24) ^
        ((word32)GetTable8(Tsbox, GETBYTE(s1, 2)) << 16) ^
        ((word32)GetTable8(Tsbox, GETBYTE(s2, 1)) <<  8) ^
        ((word32)GetTable8(Tsbox, GETBYTE(s3, 0)));
    t1 =
        ((word32)GetTable8(Tsbox, GETBYTE(s1, 3)) << 24) ^
        ((word32)GetTable8(Tsbox, GETBYTE(s2, 2)) << 16) ^
        ((word32)GetTable8(Tsbox, GETBYTE(s3, 1)) <<  8) ^
        ((word32)GetTable8(Tsbox, GETBYTE(s0, 0)));
    t2 =
        ((word32)GetTable8(Tsbox, GETBYTE(s2, 3)) << 24) ^
        ((word32)GetTable8(Tsbox, GETBYTE(s3, 2)) << 16) ^
        ((word32)GetTable8(Tsbox, GETBYTE(s0, 1)) <<  8) ^
        ((word32)GetTable8(Tsbox, GETBYTE(s1, 0)));
    t3 =
        ((word32)GetTable8(Tsbox, GETBYTE(s3, 3)) << 24) ^
        ((word32)GetTable8(Tsbox, GETBYTE(s0, 2)) << 16) ^
        ((word32)GetTable8(Tsbox, GETBYTE(s1, 1)) <<  8) ^
        ((word32)GetTable8(Tsbox, GETBYTE(s2, 0)));
    s0 = t0 ^ rk[0];
    s1 = t1 ^ rk[1];
    s2 = t2 ^ rk[2];
    s3 = t3 ^ rk[3];
#endif

    /* write out */
#ifdef LITTLE_ENDIAN_ORDER
    ByteReverseWord32(s0,&s0);
    ByteReverseWord32(s1,&s1);
    ByteReverseWord32(s2,&s2);
    ByteReverseWord32(s3,&s3);
#endif

    XMEMCPY(outBlock,                  &s0, sizeof(s0));
    XMEMCPY(outBlock +     sizeof(s0), &s1, sizeof(s1));
    XMEMCPY(outBlock + 2 * sizeof(s0), &s2, sizeof(s2));
    XMEMCPY(outBlock + 3 * sizeof(s0), &s3, sizeof(s3));
}

void AesEncrypt_C(Aes* aes, const byte* inBlock, byte* outBlock,
        word32 r)
{
    AesEncrypt_C_CUDA<<<1,1>>>(aes, inBlock, outBlock, r);
}

#if defined(HAVE_AES_ECB) && !(defined(WOLFSSL_IMX6_CAAM) && \
    !defined(NO_IMX6_CAAM_AES) && !defined(WOLFSSL_QNX_CAAM))
/* Encrypt a number of blocks using AES.
 *
 * @param [in]  aes  AES object.
 * @param [in]  in   Block to encrypt.
 * @param [out] out  Encrypted block.
 * @param [in]  sz   Number of blocks to encrypt.
 */
void AesEncryptBlocks_C(Aes* aes, const byte* in, byte* out, word32 sz)
{
    word32 i;

    for (i = 0; i < sz; i += AES_BLOCK_SIZE) {
        AesEncrypt_C(aes, in, out, aes->rounds >> 1);
        in += AES_BLOCK_SIZE;
        out += AES_BLOCK_SIZE;
    }
}
#endif

#else

/* Encrypt a block using AES.
 *
 * @param [in]  aes       AES object.
 * @param [in]  inBlock   Block to encrypt.
 * @param [out] outBlock  Encrypted block.
 * @param [in]  r         Rounds divided by 2.
 */
__global__
void AesEncrypt_C_CUDA(Aes* aes, const byte* inBlock, byte* outBlock,
        word32 r)
{
    bs_word state[AES_BLOCK_BITS];

    (void)r;

    XMEMCPY(state, inBlock, AES_BLOCK_SIZE);
    XMEMSET(((byte*)state) + AES_BLOCK_SIZE, 0, sizeof(state) - AES_BLOCK_SIZE);

    bs_encrypt(state, aes->bs_key, aes->rounds);

    XMEMCPY(outBlock, state, AES_BLOCK_SIZE);
}

void AesEncrypt_C(Aes* aes, const byte* inBlock, byte* outBlock,
        word32 r)
{
    AesEncrypt_C_CUDA<<<1,1>>>(aes, inBlock, outBlock, r);
}

#if defined(HAVE_AES_ECB) && !(defined(WOLFSSL_IMX6_CAAM) && \
    !defined(NO_IMX6_CAAM_AES) && !defined(WOLFSSL_QNX_CAAM))
/* Encrypt a number of blocks using AES.
 *
 * @param [in]  aes  AES object.
 * @param [in]  in   Block to encrypt.
 * @param [out] out  Encrypted block.
 * @param [in]  sz   Number of blocks to encrypt.
 */
void AesEncryptBlocks_C(Aes* aes, const byte* in, byte* out, word32 sz)
{
    bs_word state[AES_BLOCK_BITS];

    while (sz >= BS_BLOCK_SIZE) {
        XMEMCPY(state, in, BS_BLOCK_SIZE);
        bs_encrypt(state, aes->bs_key, aes->rounds);
        XMEMCPY(out, state, BS_BLOCK_SIZE);
        sz  -= BS_BLOCK_SIZE;
        in  += BS_BLOCK_SIZE;
        out += BS_BLOCK_SIZE;
    }
    if (sz > 0) {
        XMEMCPY(state, in, sz);
        XMEMSET(((byte*)state) + sz, 0, sizeof(state) - sz);
        bs_encrypt(state, aes->bs_key, aes->rounds);
        XMEMCPY(out, state, sz);
    }
}
#endif

#endif /* !WC_AES_BITSLICED */

#endif /* HAVE_CUDA */

#endif /* !WOLFSSL_TI_CRYPT */

